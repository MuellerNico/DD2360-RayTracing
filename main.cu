#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include <string>
#include <vector>
#include "precision_types.h"
#include "acceleration_structure.h"

#ifdef USE_OPENGL
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#endif

#define NUM_SPHERES (22 * 22 + 1 + 3)

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
#define USE_OCTREE
__device__ vec3 color(const ray& r, hitable** world, hiprandState* local_rand_state, Octree* d_octree, sphere(*d_list)[NUM_SPHERES]) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
#ifdef USE_OCTREE
        Octhit* octhit = hitTree(d_octree, r);
		// debug
        // printf("Octree hit: %d\n", octhit->num_p_hits);
		hitable_list* world_list = (hitable_list*)(*world);
		hit_record temp_rec, closest_rec;
		bool hit_anything = false;
		real_t closest_so_far = FLT_MAX;
		for(int j = 0; j < octhit->num_p_hits; j++)
		{
			int sphere_idx = octhit->possible_hits[j];
			if(sphere_idx < NUM_SPHERES) {
				if(world_list->list[sphere_idx]->hit(cur_ray, real_t(0.001f), closest_so_far, temp_rec))
				{
					hit_anything = true;
					closest_so_far = temp_rec.t;
					closest_rec = temp_rec;
				}
			}
		}
		if(hit_anything) {
            ray scattered;
            vec3 attenuation;
            if (closest_rec.mat_ptr->scatter(cur_ray, closest_rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }

#else
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
#endif
        else {
            const vec3 unit_direction = unit_vector(cur_ray.direction());
            const real_t t = real_t(0.5f) * (unit_direction.y() + (real_t)1.0f);
            const vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState* rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	// Original: Each thread gets same seed, a different sequence number, no offset
	// hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
	// BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
	// performance improvement of about 2x!
	hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, camera** cam, hitable** world, hiprandState* rand_state, Octree* d_octree, sphere(*d_list)[NUM_SPHERES]) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);
	for (int s = 0; s < ns; s++) {
		real_t u = real_t(i + hiprand_uniform(&local_rand_state)) / real_t(max_x);
		real_t v = real_t(j + hiprand_uniform(&local_rand_state)) / real_t(max_y);
		ray r = (*cam)->get_ray(u, v, &local_rand_state);
		col += color(r, world, &local_rand_state, d_octree, d_list);
	}
	
	rand_state[pixel_index] = local_rand_state;
	col /= real_t(ns);
	col[0] = sqrt(col[0]);
	col[1] = sqrt(col[1]);
	col[2] = sqrt(col[2]);
	fb[pixel_index] = col;
	
}

__global__ void render_progressive(vec3* fb, int max_x, int max_y, int current_sample, camera** cam, hitable** world, hiprandState* rand_state, Octree* d_octree, sphere(*d_list)[NUM_SPHERES]) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);

	// Render one sample per pixel per call
	real_t u = real_t(i + hiprand_uniform(&local_rand_state)) / real_t(max_x);
	real_t v = real_t(j + hiprand_uniform(&local_rand_state)) / real_t(max_y);
	ray r = (*cam)->get_ray(u, v, &local_rand_state);
	col = color(r, world, &local_rand_state, d_octree, d_list);

	rand_state[pixel_index] = local_rand_state;

	// Accumulate color
	if (current_sample == 1) {
		fb[pixel_index] = col;
	}
	else {
		fb[pixel_index] += col;
	}
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(sphere (*d_list)[NUM_SPHERES], hitable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprandState local_rand_state = *rand_state;
		(*d_list)[0] = sphere(vec3(0, -1000.0, -1), 1000,
			new lambertian(vec3(0.5, 0.5, 0.5)));	// ground plane as sphere
		int i = 1;
		for (int a = -11; a < 11; a++) {
			for (int b = -11; b < 11; b++) {
				const real_t choose_mat = RND;
				const vec3 center(a + RND, 0.2, b + RND);
				if (choose_mat < real_t(0.8f)) {
					(*d_list)[i++] = sphere(center, 0.2,
						new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
				}
				else if (choose_mat < real_t(0.95f)) {
					(*d_list)[i++] = sphere(center, 0.2,
						new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
				}
				else {
					(*d_list)[i++] = sphere(center, 0.2, new dielectric(1.5));
				}
			}
		}
		(*d_list)[i++] = sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
		(*d_list)[i++] = sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
		(*d_list)[i++] = sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
		*rand_state = local_rand_state;
		int num_hitables = 22 * 22 + 1 + 3;
		hitable** d_hitable = new hitable*[num_hitables];	// convert to array of pointers to keep changes minimal
		d_hitable[0] = &((*d_list)[0]);
		for(int i = 1; i < num_hitables; i++)
		{
			d_hitable[i] = &((*d_list)[i]);
		}
		*d_world = new hitable_list(d_hitable, num_hitables );

		const vec3 lookfrom(13, 2, 3);
		const vec3 lookat(0, 0, 0);
		const real_t dist_to_focus = 10.0; (lookfrom - lookat).length();
		const real_t aperture = 0.1;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			30.0,
			real_t(nx) / real_t(ny),
			aperture,
			dist_to_focus);
	}
}

__global__ void free_world(sphere(*d_list)[NUM_SPHERES], hitable** d_world, camera** d_camera) {
	for (int i = 0; i < NUM_SPHERES; i++) {
		delete (*d_list[i]).mat_ptr;
	}
	delete d_list;
	delete* d_world;
	delete* d_camera;
}

#ifdef USE_OPENGL
int render_in_window(const int nx, const int ny, dim3 blocks, dim3 threads, vec3* fb, camera** d_camera, hitable** d_world, hiprandState* d_rand_state, Octree* d_octree, sphere(*d_list)[NUM_SPHERES])
{
	const int num_pixels = nx * ny;

	// Initialize GLFW
	if (!glfwInit()) {
		std::cerr << "Error: GLFW initialization failed.\n";
		return -1;
	}

	// Create a GLFW window
	GLFWwindow* window = glfwCreateWindow(nx, ny, "CUDA Ray Tracer", nullptr, nullptr);
	if (!window) {
		std::cerr << "Error: Window creation failed.\n";
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);

	// Initialize GLEW (necessary to get OpenGL extensions)
	glewExperimental = GL_TRUE;
	const GLenum glew_status = glewInit();
	// Ignore GL_INVALID_ENUM error caused by glewInit()
	glGetError();
	if (glew_status != GLEW_OK) {
		std::cerr << "Error: GLEW initialization failed.\n";
		glfwDestroyWindow(window);
		glfwTerminate();
		return -1;
	}

	// Create OpenGL texture
	GLuint tex;
	glGenTextures(1, &tex);
	glBindTexture(GL_TEXTURE_2D, tex);

	// Allocate texture storage
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, nx, ny, 0, GL_RGB, GL_FLOAT, NULL);

	// Set texture parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

	int current_sample = 0;
	const int max_samples = 4000; // Set the number of samples per pixel

	// Main render loop
	while (!glfwWindowShouldClose(window) && current_sample < max_samples) {
		glfwPollEvents();

		current_sample++;

		// Launch render kernel
		render_progressive << <blocks, threads >> > (fb, nx, ny, current_sample, d_camera, d_world, d_rand_state, d_octree, d_list);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		// Copy data to OpenGL texture
		// Apply gamma correction and averaging
		std::vector<real_t> pixels(num_pixels * 3);
		for (int i = 0; i < num_pixels; ++i) {
			vec3 col = fb[i] / real_t(current_sample);
			col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2])); // Gamma correction
			pixels[i * 3 + 0] = col.r();
			pixels[i * 3 + 1] = col.g();
			pixels[i * 3 + 2] = col.b();
		}

		// Update texture
		glBindTexture(GL_TEXTURE_2D, tex);
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, nx, ny, GL_RGB, GL_FLOAT, pixels.data());

		// Render textured quad
		glClear(GL_COLOR_BUFFER_BIT);

		glEnable(GL_TEXTURE_2D);
		glBegin(GL_QUADS);
		{
			glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
			glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, -1.0f);
			glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 1.0f);
			glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, 1.0f);
		}
		glEnd();
		glDisable(GL_TEXTURE_2D);

		glfwSwapBuffers(window);

		// Optional: Display progress
		std::cout << "Sample " << current_sample << "/" << max_samples << "\r";
		std::cout.flush();
	}

	// Terminate GLFW
	glfwDestroyWindow(window);
	glfwTerminate();
}
#endif

void output_to_stream(std::ostream& ostream, const int nx, const int ny, const vec3* fb)
{
	ostream << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			const size_t pixel_index = j * nx + i;
			const int ir = static_cast<int>(255.99 * fb[pixel_index].r());
			const int ig = static_cast<int>(255.99 * fb[pixel_index].g());
			const int ib = static_cast<int>(255.99 * fb[pixel_index].b());
			ostream << ir << " " << ig << " " << ib << "\n";
		}
	}
}

void output_to_console(const int nx, const int ny, const vec3* fb)
{
	output_to_stream(std::cout, nx, ny, fb);
}

void output_to_file(const int nx, const int ny, const vec3* fb)
{
	std::ofstream outfile("output.ppm");
	output_to_stream(outfile, nx, ny, fb);
	outfile.close();
}

int main(int argc, char** argv) {
	const int nx = 1200;
	const int ny = 800;
	const int ns = 10;
	const int tx = 8;
	const int ty = 8;

	std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int output_mode = 0; // 0 = to stdout (default), 1 = disabled, 2 = to window, 3 = to file
	if (argc > 1) {
		output_mode = std::stoi(argv[1]);
	}
	std::cerr << "Output mode: " << output_mode << "\n";

	const int num_pixels = nx * ny;
	const size_t fb_size = num_pixels * sizeof(vec3);

	// allocate FB
	vec3* fb;
	checkCudaErrors(hipMallocManaged(reinterpret_cast<void**>(&fb), fb_size));

	// allocate random state
	hiprandState* d_rand_state;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_rand_state), num_pixels * sizeof(hiprandState)));
	hiprandState* d_rand_state2;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_rand_state2), 1 * sizeof(hiprandState)));

	// we need that 2nd random state to be initialized for the world creation
	rand_init << <1, 1 >> > (d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// make our world of hitables & the camera
	sphere (*d_list)[NUM_SPHERES];
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_list), NUM_SPHERES * sizeof(sphere)));
	hitable** d_world;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_world), sizeof(hitable*)));
	camera** d_camera;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_camera), sizeof(camera*)));
	create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// copy spheres to CPU to fill into octree
	// first copy the addresses on the GPU
	sphere* cpu_spheres = static_cast<sphere*>(malloc(sizeof(sphere) * NUM_SPHERES));
	checkCudaErrors(hipMemcpy(cpu_spheres, d_list, NUM_SPHERES * sizeof(sphere), hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());

	// build octree
	Octree* octree = buildOctree(cpu_spheres, NUM_SPHERES);

	// upload octree to gpu
	Octree* d_octree;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_octree),sizeof(Octree)));
	checkCudaErrors(hipMemcpy(d_octree, octree, sizeof(Octree), hipMemcpyHostToDevice));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();
	// Render our buffer
	dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
	dim3 threads(tx, ty);
	render_init << <blocks, threads >> > (nx, ny, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world, d_rand_state, d_octree, d_list);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	const double timer_seconds = static_cast<double>(stop - start) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";

	// Output methods
	switch (output_mode)
	{
	case 0:
		output_to_console(nx, ny, fb);
		break;
	case 1:
		// do nothing
		break;
#ifdef USE_OPENGL
	case 2:
		render_in_window(nx, ny, blocks, threads, fb, d_camera, d_world, d_rand_state, d_octree, d_list);
		break;
#endif
	case 3:
		output_to_file(nx, ny, fb);
	default:
		// do nothing
		break;
	}

	// clean up
	checkCudaErrors(hipDeviceSynchronize());
	free_world << <1, 1 >> > (d_list, d_world, d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(d_rand_state2));
	checkCudaErrors(hipFree(fb));

	hipDeviceReset();
}